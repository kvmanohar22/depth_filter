#include "dev_img.cuh"

using namespace learn;
using namespace std;

int main() {
  float data[] = {1, 2, 3, 4,
                5, 6, 7, 8,
                9, 10, 11, 12,
                12, 13, 14, 15};

  AbstractAllocator<float> a(4, 4);
  a.set_data(data);

  float dev2host[16];
  a.get_data(dev2host);
  for (size_t i = 0; i < 16; ++i)
    printf("i = %lu\t:\t%f\n", i, dev2host[i]);
  printf("\n");

  dim3 grid_size((4+16-1)/4, (4+16-1)/4);
  dim3 block_size(16, 16);
  // learn::test_2d<<<grid_size, block_size>>>(a.data_, a.stride_, a.width_, a.height_);
  learn::test_2d_class<<<grid_size, block_size>>>(a.ptr_);
  hipDeviceSynchronize();

  a.get_data(dev2host);
  for (size_t i = 0; i < 16; ++i)
    printf("i = %lu\t:\t%f\n", i, dev2host[i]);
  printf("\n");
}

