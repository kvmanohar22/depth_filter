#include "depth_filter/cuda/allocator.cuh"

using namespace depth_filter;
using namespace std;

int main() {
  float data[] = {1, 2, 3, 4,
                5, 6, 7, 8,
                9, 10, 11, 12,
                12, 13, 14, 15};

  AbstractAllocator<float> a(4, 4);
  a.set_data(data);

  float dev2host[16];
  a.get_data(dev2host);
  for (size_t i = 0; i < 16; ++i)
    printf("i = %lu\t:\t%f\n", i, dev2host[i]);
  printf("\n");

  dim3 grid_size((4+16-1)/4, (4+16-1)/4);
  dim3 block_size(16, 16);
  test_2d_class<<<grid_size, block_size>>>(a.ptr_);
  hipDeviceSynchronize();

  a.get_data(dev2host);
  for (size_t i = 0; i < 16; ++i)
    printf("i = %lu\t:\t%f\n", i, dev2host[i]);
  printf("\n");
}

